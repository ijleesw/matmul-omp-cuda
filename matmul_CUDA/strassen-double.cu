#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>

#include "utils.h"
#include "cudaTimer.h"


#define BLOCK_SIZE 16
#define N_TEST 50
#define DEVICE_ID 0
#define MAX_DEPTH 20


double *h_A, *h_B, *strassenRef, *cublasRef;
double *d_A[MAX_DEPTH], *d_B[MAX_DEPTH], *d_C[MAX_DEPTH];
double *d_M1[MAX_DEPTH], *d_M2[MAX_DEPTH], *d_M3[MAX_DEPTH], *d_M4[MAX_DEPTH], *d_M5[MAX_DEPTH], *d_M6[MAX_DEPTH], *d_M7[MAX_DEPTH];


template <typename ring>
void fillMatrix(ring* arr, const int N)
{
	for (int i = 0; i < N; ++i)
	{
		arr[i] = (ring) (rand() & 0xF);
	}
}


template <typename ring>
void checkResult(ring* hostRef, ring* gpuRef, const int dim, const char* name)
{
	double max_diff = 0;
	double avg_diff = 0;
	int max_idx = 0;

	for (int i = 0; i < dim*dim; ++i)
	{
		double curr_diff = abs(hostRef[i] - gpuRef[i]);
		avg_diff += curr_diff;
		if (curr_diff > max_diff)
		{
			max_diff = curr_diff;
			max_idx = i;
		}
	}
	avg_diff /= (dim*dim);

	printf("[%s] Avg difference is %.8lf. Max difference is %.8lf at index %d.\n", name, avg_diff, max_diff, max_idx);
}


template <typename ring>
__global__
void classicalMatmul(ring* A, ring* B, ring* C, const int dim)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < dim && col < dim)
	{
		ring sum = 0;
		for (int k = 0; k < dim; ++k)
		{
			sum += A[row*dim + k] * B[k*dim + col];
		}
		C[row*dim + col] = sum;
	}
}


template <typename ring>
void strassenMatmul(hipblasHandle_t& handle, ring* A, ring* B, ring* C, const int dim, const int d, const int threshold)
{
	const int dim_2 = dim/2;

	int lda = dim, ldb = dim, ldc = dim_2;
	int m = dim_2, n = dim_2;
	ring one = 1, zero = 0, m_one = -1;

	if (dim <= threshold)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid((dim+BLOCK_SIZE-1)/BLOCK_SIZE, (dim+BLOCK_SIZE-1)/BLOCK_SIZE);
		classicalMatmul<ring><<< grid, block >>>(A, B, C, dim);
		// hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &one, B, dim, A, dim, &zero, C, dim);
		return;
	}


	/* M1 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M1[d+1], dim_2, d+1, threshold);

	/* M2 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &zero, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M2[d+1], dim_2, d+1, threshold);

	/* M3 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &zero, A, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2, lda, &m_one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M3[d+1], dim_2, d+1, threshold);

	/* M4 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim+dim_2, lda, &zero, A, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &m_one, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M4[d+1], dim_2, d+1, threshold);

	/* M5 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim+dim_2, lda, &zero, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M5[d+1], dim_2, d+1, threshold);

	/* M6 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &m_one, A, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M6[d+1], dim_2, d+1, threshold);

	/* M7 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2, lda, &m_one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M7[d+1], dim_2, d+1, threshold);


	/* C1 */
	lda = dim, ldb = dim/2, ldc = dim;  // C = C + B
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C, lda, &one, d_M1[d+1], ldb, C, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, d_M4[d+1], ldb, C, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &m_one, d_M5[d+1], ldb, C, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, d_M7[d+1], ldb, C, ldc);

	/* C2 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2, lda, &one, d_M3[d+1], ldb, C+dim_2, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2, lda, &one, d_M5[d+1], ldb, C+dim_2, ldc);

	/* C3 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim, lda, &one, d_M2[d+1], ldb, C+dim_2*dim, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim, lda, &one, d_M4[d+1], ldb, C+dim_2*dim, ldc);

	/* C4 */
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim+dim_2, lda, &one, d_M1[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &m_one, d_M2[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, d_M3[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, d_M6[d+1], ldb, C+dim_2*dim+dim_2, ldc);
}


void cublasMatmul(hipblasHandle_t& handle, const double* A, const double* B, double* C, const int dim)
{
	int lda = dim, ldb = dim, ldc = dim;
	const int m = dim, n = dim, k = dim;
	const double a = 1;
	const double b = 0;
	const double *alpha = &a;
	const double *beta = &b;

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, B, ldb, A, lda, beta, C, ldc);
}


int main(int argc, char** argv)
{
	if (argc != 4)
	{
		printf("Usage: %s <dim> <threshold> <check>\n", argv[0]);
		exit(0);
	}


	/* Initialize */

	int nDim = atoi(argv[1]);
	int threshold = atoi(argv[2]);
	int check = atoi(argv[3]);

	assert(nDim >= threshold && threshold >= BLOCK_SIZE);

	setDeviceAndGetInfo(DEVICE_ID);

	size_t nBytes = nDim * nDim * sizeof(double);

	h_A = (double*) malloc(nBytes);
	h_B = (double*) malloc(nBytes);
	strassenRef = (double*) malloc(nBytes);
	cublasRef = (double*) malloc(nBytes);

	srand(0);
	fillMatrix<double>(h_A, nDim*nDim);
	fillMatrix<double>(h_B, nDim*nDim);

	int depth, _dim = nDim;
	for (depth = 0; depth < MAX_DEPTH && _dim > 0; ++depth)
	{
		hipMalloc((double**) &d_A[depth], _dim*_dim*sizeof(double));
		hipMalloc((double**) &d_B[depth], _dim*_dim*sizeof(double));

		if (depth == 0)
		{
			hipMalloc((double**) &d_C[depth], _dim*_dim*sizeof(double));
		}
		else
		{
			hipMalloc((double**) &d_M1[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M2[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M3[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M4[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M5[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M6[depth], _dim*_dim*sizeof(double));
			hipMalloc((double**) &d_M7[depth], _dim*_dim*sizeof(double));
		}
		_dim /= 2;
	}

	hipMemcpy(d_A[0], h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B[0], h_B, nBytes, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);


	/* Run strassenMatmul */

	CudaTimer ct;

	ct.start();
	for (int i = 0; i < N_TEST; ++i)
	{
		strassenMatmul<double>(handle, d_A[0], d_B[0], d_C[0], nDim, 0, threshold);
	}
	ct.stop();

	hipMemcpy(strassenRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
	printf("[strassenMatmul] %.5fms\n", ct.value()/N_TEST);


	/* Check with cuBLAS */

	if (check)
	{
		cublasMatmul(handle, d_A[0], d_B[0], d_C[0], nDim);
		hipMemcpy(cublasRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
		checkResult<double>(cublasRef, strassenRef, nDim, "strassenMatmul");
	}


	/* Free memory */

	hipblasDestroy(handle);

	for (int i = 0; i < depth; ++i)
	{
		hipFree(d_A[i]);
		hipFree(d_B[i]);

		if (i == 0)
		{
			hipFree(d_C[i]);
		}
		else
		{
			hipFree(d_M1[i]);
			hipFree(d_M2[i]);
			hipFree(d_M3[i]);
			hipFree(d_M4[i]);
			hipFree(d_M5[i]);
			hipFree(d_M6[i]);
			hipFree(d_M7[i]);
		}
	}

	hipDeviceReset();

	free(h_A);
	free(h_B);
	free(strassenRef);
	free(cublasRef);

	printf("Done.\n");

	return 0;
}
