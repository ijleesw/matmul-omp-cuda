#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>

#include "utils.h"
#include "cudaTimer.h"


#define BLOCK_SIZE 16
#define N_TEST 50
#define DEVICE_ID 0
#define MAX_DEPTH 20


hipFloatComplex *h_A, *h_B, *matmulRef, *cublasRef;
hipFloatComplex *d_A[MAX_DEPTH], *d_B[MAX_DEPTH], *d_C[MAX_DEPTH];
hipFloatComplex *d_M1[MAX_DEPTH], *d_M2[MAX_DEPTH], *d_M3[MAX_DEPTH], *d_M4[MAX_DEPTH], *d_M5[MAX_DEPTH], *d_M6[MAX_DEPTH], *d_M7[MAX_DEPTH];


template <typename ring>
void fillMatrix(ring* arr, const int N)
{
	for (int i = 0; i < N; ++i)
	{
		arr[i].x = (rand() & 0xFF) / 10;
		arr[i].y = (rand() & 0xFF) / 10;
	}
}


template <typename ring>
void checkResult(ring* lhs, ring* rhs, const int dim, const char* name)
{
	double max_diff = 0;
	double avg_diff = 0;
	int max_idx = 0;

	for (int i = 0; i < dim*dim; ++i)
	{
		double curr_diff = hipCabsf(hipCsubf(lhs[i], rhs[i]));
		avg_diff += curr_diff;
		if (curr_diff > max_diff)
		{
			max_diff = curr_diff;
			max_idx = i;
		}
	}
	avg_diff /= (dim*dim);

	printf("[%s] Avg diff is %.8lf. Max diff is %.8lf at index %d.\n", name, avg_diff, max_diff, max_idx);
}


template <typename ring>
__global__
void classicalMatmul(ring* A, ring* B, ring* C, const int dim)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int gd = gridDim.x;

	__shared__ ring _A[BLOCK_SIZE][BLOCK_SIZE], _B[BLOCK_SIZE][BLOCK_SIZE];

	if (row < dim && col < dim)
	{
		ring sum = make_hipFloatComplex(0, 0);
		for (int k = 0; k < gd; ++k)
		{
			_A[threadIdx.y][threadIdx.x] = A[row*dim + k*BLOCK_SIZE + threadIdx.x];
			_B[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE+threadIdx.y) * dim + col];
			__syncthreads();

			for (int l = 0; l < BLOCK_SIZE; ++l)
			{
				sum = hipCaddf(sum, hipCmulf(_A[threadIdx.y][l], _B[l][threadIdx.x]));
			}
			__syncthreads();
		}

		C[row*dim + col] = sum;
	}
}


template <typename ring>
void strassenMatmul(hipblasHandle_t& handle, ring* A, ring* B, ring* C, const int dim, const int d, const int threshold)
{
	const int dim_2 = dim/2;

	int lda = dim, ldb = dim, ldc = dim_2;
	int m = dim_2, n = dim_2;
	ring one = make_hipFloatComplex(1, 0);
	ring zero = make_hipFloatComplex(0, 0);
	ring m_one = make_hipFloatComplex(-1, 0);

	if (dim <= threshold)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid((dim+BLOCK_SIZE-1)/BLOCK_SIZE, (dim+BLOCK_SIZE-1)/BLOCK_SIZE);
		classicalMatmul<ring><<< grid, block >>>(A, B, C, dim);
		// hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, &one, B, dim, A, dim, &zero, C, dim);
		return;
	}


	/* M1 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M1[d+1], dim_2, d+1, threshold);

	/* M2 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &zero, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M2[d+1], dim_2, d+1, threshold);

	/* M3 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &zero, A, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2, lda, &m_one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M3[d+1], dim_2, d+1, threshold);

	/* M4 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim+dim_2, lda, &zero, A, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &m_one, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M4[d+1], dim_2, d+1, threshold);

	/* M5 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim+dim_2, lda, &zero, B, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M5[d+1], dim_2, d+1, threshold);

	/* M6 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &m_one, A, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M6[d+1], dim_2, d+1, threshold);

	/* M7 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2, lda, &m_one, A+dim_2*dim+dim_2, ldb, d_A[d+1], ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &one, B+dim_2*dim+dim_2, ldb, d_B[d+1], ldc);
	strassenMatmul(handle, d_A[d+1], d_B[d+1], d_M7[d+1], dim_2, d+1, threshold);


	/* C1 */
	lda = dim, ldb = dim/2, ldc = dim;  // C = C + B
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C, lda, &one, d_M1[d+1], ldb, C, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, d_M4[d+1], ldb, C, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &m_one, d_M5[d+1], ldb, C, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, d_M7[d+1], ldb, C, ldc);

	/* C2 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2, lda, &one, d_M3[d+1], ldb, C+dim_2, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2, lda, &one, d_M5[d+1], ldb, C+dim_2, ldc);

	/* C3 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim, lda, &one, d_M2[d+1], ldb, C+dim_2*dim, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim, lda, &one, d_M4[d+1], ldb, C+dim_2*dim, ldc);

	/* C4 */
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim+dim_2, lda, &one, d_M1[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &m_one, d_M2[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, d_M3[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, d_M6[d+1], ldb, C+dim_2*dim+dim_2, ldc);
}


void cublasMatmul(hipblasHandle_t& handle, const hipFloatComplex* A, const hipFloatComplex* B, hipFloatComplex* C, const int dim)
{
	int lda = dim, ldb = dim, ldc = dim;
	const int m = dim, n = dim, k = dim;
	const hipFloatComplex a = make_hipFloatComplex(1, 0);
	const hipFloatComplex b = make_hipFloatComplex(0, 0);
	const hipFloatComplex *alpha = &a;
	const hipFloatComplex *beta = &b;

	hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, B, ldb, A, lda, beta, C, ldc);
}


int main(int argc, char** argv)
{
	if (argc != 4)
	{
		printf("Usage: %s <dim> <threshold> <check>\n", argv[0]);
		exit(0);
	}


	/* Initialize */

	int nDim = atoi(argv[1]);
	int threshold = atoi(argv[2]);
	int check = atoi(argv[3]);

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((nDim+BLOCK_SIZE-1)/BLOCK_SIZE, (nDim+BLOCK_SIZE-1)/BLOCK_SIZE);

	assert(nDim >= threshold && threshold >= BLOCK_SIZE);

	setDeviceAndGetInfo(DEVICE_ID);

	size_t nBytes = nDim * nDim * sizeof(hipFloatComplex);

	h_A = (hipFloatComplex*) malloc(nBytes);
	h_B = (hipFloatComplex*) malloc(nBytes);
	matmulRef = (hipFloatComplex*) malloc(nBytes);
	cublasRef = (hipFloatComplex*) malloc(nBytes);

	srand(0);
	fillMatrix<hipFloatComplex>(h_A, nDim*nDim);
	fillMatrix<hipFloatComplex>(h_B, nDim*nDim);

	int depth, _dim = nDim;
	for (depth = 0; depth < MAX_DEPTH && _dim > 0; ++depth)
	{
		hipMalloc((hipFloatComplex**) &d_A[depth], _dim*_dim*sizeof(hipFloatComplex));
		hipMalloc((hipFloatComplex**) &d_B[depth], _dim*_dim*sizeof(hipFloatComplex));

		if (depth == 0)
		{
			hipMalloc((hipFloatComplex**) &d_C[depth], _dim*_dim*sizeof(hipFloatComplex));
		}
		else
		{
			hipMalloc((hipFloatComplex**) &d_M1[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M2[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M3[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M4[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M5[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M6[depth], _dim*_dim*sizeof(hipFloatComplex));
			hipMalloc((hipFloatComplex**) &d_M7[depth], _dim*_dim*sizeof(hipFloatComplex));
		}
		_dim /= 2;
	}

	hipMemcpy(d_A[0], h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B[0], h_B, nBytes, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	CudaTimer ct;


	/* Prepare result */

	if (check)
	{
		cublasMatmul(handle, d_A[0], d_B[0], d_C[0], nDim);
		hipMemcpy(cublasRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
	}


	/* Run classicalMatmul */

	ct.start();
	for (int i = 0; i < N_TEST; ++i)
	{
		classicalMatmul<hipFloatComplex><<< grid, block >>>(d_A[0], d_B[0], d_C[0], nDim);
		hipDeviceSynchronize();
	}
	ct.stop();
	printf("[classicalMatmul] %.5fms\n", ct.value()/N_TEST);

	if (check)
	{
		hipMemcpy(matmulRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
		checkResult<hipFloatComplex>(cublasRef, matmulRef, nDim, "classicalMatmul");
	}


	/* Run strassenMatmul */

	ct.start();
	for (int i = 0; i < N_TEST; ++i)
	{
		strassenMatmul<hipFloatComplex>(handle, d_A[0], d_B[0], d_C[0], nDim, 0, threshold);
	}
	ct.stop();

	hipMemcpy(matmulRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
	printf("[strassenMatmul] %.5fms\n", ct.value()/N_TEST);

	if (check)
	{
		hipMemcpy(matmulRef, d_C[0], nBytes, hipMemcpyDeviceToHost);
		checkResult<hipFloatComplex>(cublasRef, matmulRef, nDim, "strassenMatmul");
	}


	/* Free memory */

	hipblasDestroy(handle);

	for (int i = 0; i < depth; ++i)
	{
		hipFree(d_A[i]);
		hipFree(d_B[i]);

		if (i == 0)
		{
			hipFree(d_C[i]);
		}
		else
		{
			hipFree(d_M1[i]);
			hipFree(d_M2[i]);
			hipFree(d_M3[i]);
			hipFree(d_M4[i]);
			hipFree(d_M5[i]);
			hipFree(d_M6[i]);
			hipFree(d_M7[i]);
		}
	}

	hipDeviceReset();

	free(h_A);
	free(h_B);
	free(matmulRef);
	free(cublasRef);

	printf("Done.\n");

	return 0;
}
